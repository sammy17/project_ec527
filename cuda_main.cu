
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <math.h>
#include <time.h>


// Assertion to check for errors
#define CUDA_SAFE_CALL(ans) { gpuAssert((ans), (char *)__FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
  if (code != hipSuccess)
  {
    fprintf(stderr, "CUDA_SAFE_CALL: %s %s %d\n",
                                       hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}

#define RED  "\x1B[31m"
#define RESET  "\x1B[0m"
#define PRINT_TIME
#define PRINT_TIME1
//#define USE_RANDOM

#ifdef USE_RANDOM
#define INPUT_DIMENSIONS 1000
#else
#define INPUT_DIMENSIONS 28
#endif

// #define INPUT_DIMENSIONS 28
#define C1_LENGTH 6
#define C1_DIMENSIONS 24
#define S1_LENGTH 6
#define S1_DIMENSIONS 12
#define C2_LENGTH 12
#define C2_DIMENSIONS 8
#define S2_LENGTH 12
#define S2_DIMENSIONS 4
#define OUTPUT_LENGTH 10
#define KERNEL_SIZE 5
#define UPPER 1
#define LOWER -1
#define BATCH_SIZE 200
#define LEARNING_RATE 0.01



// struct VECTOR{
//     double* vector;
// };
//typedef struct VECTOR vector;
typedef double* vector;

vector new_vector(int length){
    vector n = (double*)malloc(sizeof(double)*length);
    //new->vector = malloc(sizeof(double)*length);
    return n;
}

double* cu_memcpy_vector(double* h_vec, int size){
	double* d_vec;
	hipMalloc((void**) &(d_vec), sizeof(double)*size);
	hipMemcpy(d_vec, h_vec, sizeof(double)*size, hipMemcpyHostToDevice);
	return d_vec;
}

double* cu_malloc_vector(int size){
	double* d_vec;
	hipMalloc((void**) &(d_vec), sizeof(double)*size);
	return d_vec;
}



// struct ARRAY{
//     double** matrix;
// };
//typedef struct ARRAY array;
typedef double** array;

array new_array(int rows, int columns){
    //array* new = malloc(sizeof(array*));
    array n = (double**)malloc(sizeof(double*)*rows);
    int i;
    for(i = 0; i < rows; i++){
        n[i] = (double*)malloc(sizeof(double)*columns);
    }
    return n;
}

double* cu_memcpy_array(double** h_ar, int rows, int cols){
	double* d_ar;
	int i,ret;
	ret = hipMalloc((void**) &(d_ar), sizeof(double)*(rows*cols));
	if (ret!=0){
		printf("ERROR: hipMalloc failed\n");
		exit(1);	
	}
	for (i = 0; i<rows; i++){
		hipMemcpy(d_ar, h_ar[i], sizeof(double)*cols, hipMemcpyHostToDevice); //cols=size of one row
		d_ar += cols;
	}
	return d_ar;
}

double* cu_malloc_array(int rows, int cols){
	double* d_ar;
	hipMalloc((void**) &(d_ar), sizeof(double)*(rows*cols));
	return d_ar;
}

#define IMUL(a, b) __mul24(a, b)

__global__ void c1_kernel (double* input_image, double* kernels, double* biases, double* output_images) {

	__shared__ double kernel[KERNEL_SIZE][KERNEL_SIZE];		//5x5 kernel
	
  	//__shared__ double ouput_image[C1_DIMENSIONS][C1_DIMENSIONS];	//24x24 output image

	const int col = IMUL(blockIdx.x, C1_DIMENSIONS) + threadIdx.x;
  	const int row = IMUL(blockIdx.y, C1_DIMENSIONS) + threadIdx.y;
  	const int tx = threadIdx.x;
  	const int ty = threadIdx.y;
	const int bx = blockIdx.x;
	const int by = blockIdx.y;
	const double bias = biases[by];

	int i, j;
	double sum = 0.0;
	double total = 0.0;
	if (tx<KERNEL_SIZE && ty<KERNEL_SIZE){ // divide copying to shared memory among threads
		kernel[ty][tx] = kernels[by*KERNEL_SIZE*KERNEL_SIZE+ty*KERNEL_SIZE+tx]; // get the relevant kernel element from the kernel array
	}
	__syncthreads();

	for(i=0; i<KERNEL_SIZE; i++){
		for(j=0; j<KERNEL_SIZE; j++){
			sum += input_image[row*INPUT_DIMENSIONS+col+(i*KERNEL_SIZE+j)] * kernel[i][j];
		}
	}
	total = sum + bias;
	if (total<0){
		total = (0.01*total);
	}
	//total = 0;
	output_images[row*INPUT_DIMENSIONS+col] = total;
	//__syncthreads();

}

// struct KERNEL_ARRAY{
//     array** kernels;
// };
// typedef struct KERNEL_ARRAY kernel_array;
typedef array** kernel_array;

kernel_array new_kernel_array(int rows, int columns, int kernel_dimensions){
    kernel_array n = (double****)malloc(sizeof(array**)*rows); //malloc(sizeof(kernel_array*)); //
    // new->kernels = malloc(sizeof(array**)*rows);
    int j,k;
    for(j = 0; j < rows; j++){
        n[j] = (double***)malloc(sizeof(array*)*columns);
    }
    for(j = 0; j < rows; j++){
        for(k = 0; k < columns; k++){
            n[j][k] = new_array(kernel_dimensions,kernel_dimensions);
        }
    }
    return n;
}

double* cu_memcpy_kernel(double**** h_ker, int rows, int cols, int kernel_d){
	double* d_ker;
	int i,j,k;
	hipMalloc((void**) &(d_ker), sizeof(double)*(rows*cols*kernel_d*kernel_d));
	for (i = 0; i<rows; i++){
		for (j = 0; j<cols; j++){
			for (k = 0; k<kernel_d; k++){
				hipMemcpy(d_ker, h_ker[i][j][k], sizeof(double)*kernel_d, hipMemcpyHostToDevice); //cols=size of one row
				d_ker += kernel_d;
			}
		}
	}
	return d_ker;
}

/*
double* cu_memcpy_kernel_col(double**** h_ker, int rows, int col, int kernel_d){
        double* d_ker;
        int i,j,k;
        cudaMalloc((void**) &(d_ker), sizeof(double)*(rows*cols*kernel_d*kernel_d));
        for (i = 0; i<rows; i++){
                for (k = 0; k<kernel_d; k++){
                	cudaMemcpy(d_ker, h_ker[i][col][k], sizeof(double)*kernel_d, cudaMemcpyHostToDevice); //cols=size of one row
                        d_ker += kernel_d;
                }
        }
        return d_ker;
}
*/

double* cu_malloc_kernel(int rows, int cols, int kernel_d){
	double* d_ker;
	hipMalloc((void**) &(d_ker), sizeof(double)*(rows*cols*kernel_d*kernel_d));
	return d_ker;
}

struct IMAGE_VECTOR{
    array* image;
};
typedef struct IMAGE_VECTOR image_vector;

image_vector* new_image_vector(int image_dimensions, int length){
    image_vector* n = (image_vector*)malloc(sizeof(image_vector*));
    n->image = (double***)malloc(sizeof(array*)*length);
    int i;
    for(i = 0; i < length; i++){
        n->image[i] = new_array(image_dimensions, image_dimensions);
    }
    return n;
}

double* cu_memcpy_image(image_vector* h_img, int img_dim, int length){
	double* d_img;
	int i,j,k;
	hipMalloc((void**) &(d_img), sizeof(double)*(img_dim*img_dim*length));
	for (i = 0; i<length; i++){
		for (j = 0; j<img_dim; j++){
			hipMemcpy(d_img, h_img->image[i][j], sizeof(double)*img_dim, hipMemcpyHostToDevice); //cols=size of one row
			d_img += img_dim;
		}
	}
	return d_img;
}

image_vector* h_memcpy_image(double* d_img, int img_dim, int length){
	int i,j;
	image_vector* h_img = new_image_vector(img_dim,length);
	//h_x = (double *) malloc();

        for (i = 0; i<length; i++){
                for (j = 0; j<img_dim; j++){
                        hipMemcpy(h_img->image[i][j], d_img, sizeof(double)*img_dim, hipMemcpyDeviceToHost); //cols=size of one row
                        d_img += img_dim;
                }
        }
	return h_img;
}

double* cu_malloc_images(int img_dim, int length){
	double* d_img;
	hipMalloc((void**) &(d_img), sizeof(double)*(img_dim*img_dim*length));
	return d_img;
}

struct CNN{
    array input_image;
    kernel_array C1_Kernels;
    vector C1_Biases;
    image_vector* C1_Images;
    image_vector* S1_Images;
    kernel_array C2_Kernels;
    vector C2_Biases;
    image_vector* C2_Images;
    image_vector* S2_Images;
    vector S2_Vector;
    array output_weights;
    vector output_biases;
    vector calculated_output;
    vector desired_output;
};
typedef struct CNN cnn;

struct CUDA_CNN{
    double* input_image;
    double* C1_Kernels;
    double* C1_Biases;
    double* C1_Images;
    double* S1_Images;
    double* C2_Kernels;
    double* C2_Biases;
    double* C2_Images;
    double* S2_Images;
    double* S2_Vector;
    double* output_weights;
    double* output_biases;
    double* calculated_output;
    double* desired_output;
};
typedef struct CUDA_CNN cucnn;

cnn* new_cnn(){
    cnn* n = (cnn*)malloc(sizeof(cnn));
    // new->input_image = malloc(sizeof(array*));
    // new->C1_Kernels = malloc(sizeof(kernel_array*));
    //new->C1_Biases = malloc(sizeof(vector*));
    // new->C1_Images = malloc(sizeof(image_vector)); chath
    // new->S1_Images = malloc(sizeof(image_vector));
    // new->C2_Kernels = malloc(sizeof(kernel_array*));
    //new->C2_Biases = malloc(sizeof(vector*));
    // new->C2_Images = malloc(sizeof(image_vector*)); chath
    // new->S2_Images = malloc(sizeof(image_vector*));
    //new->S2_Vector = malloc(sizeof(vector*));
    // new->output_weights = malloc(sizeof(array*));
    //new->output_biases = malloc(sizeof(vector*));
    //new->calculated_output = malloc(sizeof(vector*));
    //new->desired_output = malloc(sizeof(vector*));
    n->input_image = new_array(INPUT_DIMENSIONS, INPUT_DIMENSIONS);
    n->C1_Kernels = new_kernel_array(1, C1_LENGTH, KERNEL_SIZE);
    n->C1_Biases = new_vector(C1_LENGTH);
    n->C1_Images = new_image_vector(C1_DIMENSIONS, C1_LENGTH);
    n->S1_Images = new_image_vector(S1_DIMENSIONS, S1_LENGTH);
    n->C2_Kernels = new_kernel_array(S1_LENGTH, C2_LENGTH, KERNEL_SIZE);
    n->C2_Biases = new_vector(C2_LENGTH);
    n->C2_Images = new_image_vector(C2_DIMENSIONS, C2_LENGTH);
    n->S2_Images = new_image_vector(S2_DIMENSIONS, S2_LENGTH);
    n->S2_Vector = new_vector(S2_DIMENSIONS * S2_DIMENSIONS * S2_LENGTH);
    n->output_weights = new_array(OUTPUT_LENGTH, S2_DIMENSIONS * S2_DIMENSIONS * S2_LENGTH);
    n->output_biases = new_vector(OUTPUT_LENGTH);
    n->calculated_output = new_vector(OUTPUT_LENGTH);
    n->desired_output = new_vector(OUTPUT_LENGTH);
    return n;
}

cucnn* cu_memcpy_cnn(cnn network){
	cucnn* cu_network;
	hipMalloc((void**) &(cu_network), sizeof(cu_network));
	cu_network->input_image = cu_memcpy_array(network.input_image, INPUT_DIMENSIONS, INPUT_DIMENSIONS);
	cu_network->C1_Kernels = cu_memcpy_kernel(network.C1_Kernels, 1, C1_LENGTH, KERNEL_SIZE);
	cu_network->C1_Biases = cu_memcpy_vector(network.C1_Biases, C1_LENGTH);
	cu_network->C1_Images = cu_memcpy_image(network.C1_Images, C1_DIMENSIONS, C1_LENGTH);
	cu_network->S1_Images = cu_memcpy_image(network.S1_Images, S1_DIMENSIONS, S1_LENGTH);
	cu_network->C2_Kernels = cu_memcpy_kernel(network.C2_Kernels, S1_LENGTH, C2_LENGTH, KERNEL_SIZE);
	cu_network->C2_Biases = cu_memcpy_vector(network.C2_Biases, C2_LENGTH);
	cu_network->C2_Images = cu_memcpy_image(network.C2_Images, C2_DIMENSIONS, C2_LENGTH);
	cu_network->S2_Images = cu_memcpy_image(network.S2_Images, S2_DIMENSIONS, S2_LENGTH);
	cu_network->S2_Vector = cu_memcpy_vector(network.S2_Vector, S2_DIMENSIONS * S2_DIMENSIONS * S2_LENGTH);
	cu_network->output_weights = cu_memcpy_array(network.output_weights, OUTPUT_LENGTH, S2_DIMENSIONS * S2_DIMENSIONS * S2_LENGTH);
	cu_network->output_biases = cu_memcpy_vector(network.output_biases, OUTPUT_LENGTH);
	cu_network->calculated_output = cu_memcpy_vector(network.calculated_output, OUTPUT_LENGTH);
	cu_network->desired_output = cu_memcpy_vector(network.desired_output, OUTPUT_LENGTH);
	return cu_network;
}

/*
__global__ void kernel_bp (cucnn network, ) {

  const int col = IMUL(blockIdx.x, BLOCK_X) + threadIdx.x;
  const int row = IMUL(blockIdx.y, BLOCK_Y) + threadIdx.y;

  int i;
  float sum = 0;

  for(i = 0; i < ARRAY_LEN; i += 1) {
      sum += (m[row * ARRAY_LEN + i] * n[i * ARRAY_LEN + col]);
  }
  result[row * ARRAY_LEN + col] = sum; 
}
*/

void load_kernels(cnn* network);
void load_network(cnn* network);
void load_weights(cnn* network);
void load_biases(cnn* network);
void load_output(cnn* network);
double uniform_rand();
void zero_activations(cnn* network);
void zero_parameters(cnn* network);
void zero_network(cnn* network);
void read_headers(FILE *image_ptr, FILE *label_ptr);
void load_image(cnn* network, FILE *image_ptr, FILE *label_ptr);
void load_C1(cnn* network);
void load_C1_gpu(cnn* network);
void load_S1(cnn* network);
void load_C2(cnn* network);
void load_S2(cnn* network);
void concatenate_S2(cnn* network);
void load_output(cnn* network);
double dot_product(array array1, array array2, int dimensions);
double activation(double x);
double activation_derivative(double x);
void free_array(array array, int rows);
void free_image_vector(image_vector* images, int dimensions, int length);
double average_matrix(array image, int dimensions);
void backpropagation(cnn* network, cnn* gradient);
void forward_propagate(cnn* network);
void gradient_descent(cnn* network, cnn* gradient);
double loss_function(cnn* network);
void print_output(cnn* network);
void update_batch_gradient(cnn* image_gradient, cnn* batch_gradient);
void print_image(array image, int dimensions);
double output_activation(double x);
double output_activation_derivative(double x);
void save_parameters(cnn* network);
void save_kernels(cnn* network);
void save_weights(cnn* network);
void save_biases(cnn* network);

double interval(struct timespec start, struct timespec end)
{
  struct timespec temp;
  temp.tv_sec = end.tv_sec - start.tv_sec;
  temp.tv_nsec = end.tv_nsec - start.tv_nsec;
  if (temp.tv_nsec < 0) {
    temp.tv_sec = temp.tv_sec - 1;
    temp.tv_nsec = temp.tv_nsec + 1000000000;
  }
  return (((double)temp.tv_sec) + ((double)temp.tv_nsec)*1.0e-9);
}

double fRand(void)
{
  double f = (double)random() / (double)(RAND_MAX);
  return f;
}

int main(int argc, char** argv) {
    FILE *image_ptr;
    FILE *label_ptr;
    srand(time(NULL));
    struct timespec time_start, time_stop;
    struct timespec time_start1, time_stop1;
    double bp_time, total_time;
    cnn* network = new_cnn();
    cnn* batch_gradient = new_cnn();
    cnn* image_gradient = new_cnn();
    
    zero_network(batch_gradient);
    zero_network(image_gradient);
    zero_network(network);
    load_network(network);
    int n, i, j, m, N, M;
    N = 1; M = 1; // M = epoch, N = iterations
    double time_stamp[M][N][BATCH_SIZE];
    
    //Training Loop
    double batch_loss = 0;
    double iteration_loss = 0;
    clock_gettime(CLOCK_REALTIME, &time_start1);
    for(m = 0; m < M; m++){ // EPOCH
        image_ptr = fopen("Images.idx3-ubyte","r");
        label_ptr = fopen("Labels.idx1-ubyte", "r");
        read_headers(image_ptr, label_ptr);
        for(n = 0; n < N; n++){ // Images per batch = (60000)/BATCH_SIZE //(double)(60000)/BATCH_SIZE - N was 60000/BATCH_SIZE
            zero_network(batch_gradient);
            for(i = 0; i < BATCH_SIZE; i++){
                zero_network(image_gradient);
                zero_activations(network);
                load_image(network, image_ptr, label_ptr);
                forward_propagate(network);
                clock_gettime(CLOCK_REALTIME, &time_start);
                backpropagation(network, image_gradient);
                clock_gettime(CLOCK_REALTIME, &time_stop);
                time_stamp[m][n][i] = interval(time_start, time_stop);
                update_batch_gradient(image_gradient, batch_gradient);
                batch_loss += loss_function(network);
            }
            gradient_descent(network, batch_gradient);
            printf("Loss for batch(%d, %d): %lf\n",m, n, batch_loss/BATCH_SIZE);
            iteration_loss += batch_loss;
            batch_loss = 0;
            save_parameters(network);
        }
        printf("Loss for iteration(%d): %lf\n",m,iteration_loss/60000);
        iteration_loss = 0;
        fclose(image_ptr);
        fclose(label_ptr);   
    }
    clock_gettime(CLOCK_REALTIME, &time_stop1);
    for(m = 0; m < M; m++){
        for(n = 0; n < N; n++){
            for(i = 0; i < BATCH_SIZE; i++){
                bp_time += time_stamp[m][n][i];
            }
        }
    }
    printf("Average time for backpropagation over %d iterations = %lf s\n",M*N*BATCH_SIZE, (bp_time/(M*N*BATCH_SIZE)));
    printf("Total time for backpropagation = %lf s\n",bp_time);
    total_time = interval(time_start1, time_stop1);
    printf("Total elapsed time = %lf s\n",total_time);

    return (EXIT_SUCCESS);
}

void save_parameters(cnn* network){
    save_kernels(network);
    save_weights(network);
    save_biases(network);
}

void save_kernels(cnn* network){
    FILE *kernels_ptr = fopen("kernels","w");
    int i, n, j, k;
    
    for(i = 0; i < C1_LENGTH; i++){
        for(j = 0; j < KERNEL_SIZE; j++){
            for(k = 0; k < KERNEL_SIZE; k++){
                fprintf(kernels_ptr,"%lf ",network->C1_Kernels[0][i][j][k]);
            }
        }
    }
    
    for(i = 0; i < S1_LENGTH; i++){
        for(n = 0; n < C2_LENGTH; n++){
            for(j = 0; j < KERNEL_SIZE; j++){
                for(k = 0; k < KERNEL_SIZE; k++){
                    fprintf(kernels_ptr,"%lf ",network->C2_Kernels[i][n][j][k]);
                }
            }
        }
    }
    fclose(kernels_ptr);
}

void save_weights(cnn* network){
    FILE *weight_ptr = fopen("weights","w");
    int j, k;
    for(j = 0; j < OUTPUT_LENGTH; j++){
        for(k = 0; k < S2_LENGTH * S2_DIMENSIONS * S2_DIMENSIONS; k++){
            fprintf(weight_ptr,"%lf ",network->output_weights[j][k]);
        }
    }
    fclose(weight_ptr);
}

void save_biases(cnn* network){
    FILE *bias_ptr = fopen("biases","w");
    int i;
    for(i = 0; i < C1_LENGTH; i++){
        fprintf(bias_ptr,"%lf ",network->C1_Biases[i]);
    }
    for(i = 0; i < C2_LENGTH; i++){
        fprintf(bias_ptr,"%lf ",network->C2_Biases[i]);
    }
    for(i = 0; i < OUTPUT_LENGTH; i++){
        fprintf(bias_ptr,"%lf ",network->output_biases[i]);
    }
    fclose(bias_ptr);
}

void print_output(cnn* network){
    int i;
    for(i = 0; i < OUTPUT_LENGTH; i++){
        printf("%lf == %lf\n",network->calculated_output[i],network->desired_output[i]);
    }
    printf("\n");
}

void forward_propagate(cnn* network){
    //load_C1(network);
    load_C1_gpu(network);
    load_S1(network);
    load_C2(network);
    load_S2(network);
    concatenate_S2(network);
    load_output(network);
}

//Loads the network's weights, kernels, and biases
void load_network(cnn* network){
    load_kernels(network);
    load_weights(network);
    load_biases(network);
}

//Loads a uniform distribution of values for the kernels in the network
void load_kernels(cnn* network){
    int i, n, j, k;
    FILE *kernel_ptr = fopen("kernels","r");
    double value = 0;
    //C1 will have C1_LENGTH, KERNEL_SIZE x KERNEL_SIZE kernels
    for(i = 0; i < C1_LENGTH; i++){
        for(j = 0; j < KERNEL_SIZE; j++){
            for(k = 0; k < KERNEL_SIZE; k++){
                fscanf(kernel_ptr,"%lf",&value);
                network->C1_Kernels[0][i][j][k] = value;
            }
        }
        //print_image(network->C1_Kernels->kernels[0][i], KERNEL_SIZE);
    }
    
    //C2 will have C2_LENGTH, KERNEL_SIZE x KERNEL_SIZE kernels
    for(i = 0; i < S1_LENGTH; i++){
        for(n = 0; n < C2_LENGTH; n++){
            for(j = 0; j < KERNEL_SIZE; j++){
                for(k = 0; k < KERNEL_SIZE; k++){
                    fscanf(kernel_ptr,"%lf",&value);
                    network->C2_Kernels[i][n][j][k] = value;
                }
            }
            //print_image(network->C2_Kernels->kernels[i][n], KERNEL_SIZE);
        }
    }
    fclose(kernel_ptr);
}

//Loads a uniform distribution of values for the weights in the network
void load_weights(cnn* network){
    FILE *weight_ptr = fopen("weights","r");
    int j, k;
    double value = 0;
    for(j = 0; j < OUTPUT_LENGTH; j++){
        for(k = 0; k < S2_LENGTH * S2_DIMENSIONS * S2_DIMENSIONS; k++){
            fscanf(weight_ptr,"%lf",&value);
            network->output_weights[j][k] = value;
        }
    }
    fclose(weight_ptr);
}

//Sets all biases to 0
void load_biases(cnn* network){
    int i;
    FILE *biases_ptr = fopen("biases","r");
    double value = 0;
    for(i = 0; i < C1_LENGTH; i++){
        fscanf(biases_ptr,"%lf",&value);
        network->C1_Biases[i] = value;
    }
    for(i = 0; i < C2_LENGTH; i++){
        fscanf(biases_ptr,"%lf",&value);
        network->C2_Biases[i] = value;
    }
    for(i = 0; i < OUTPUT_LENGTH; i++){
        fscanf(biases_ptr,"%lf",&value);
        network->output_biases[i] = value;
    }
    fclose(biases_ptr);
}

//Sets all values within a network to 0
void zero_network(cnn* network){
    zero_activations(network);
    zero_parameters(network);
}

//Sets all activations within a network to 0
void zero_activations(cnn* network){
    int i, j, k;
    for(i = 0; i < C1_LENGTH; i++){
        for(j = 0; j < C1_DIMENSIONS; j++){
            for(k = 0; k < C1_DIMENSIONS; k++){
                network->C1_Images->image[i][j][k] = 0;
            }
        }
    }
    for(i = 0; i < S1_LENGTH; i++){
        for(j = 0; j < S1_DIMENSIONS; j++){
            for(k = 0; k < S1_DIMENSIONS; k++){
                network->S1_Images->image[i][j][k] = 0;
            }
        }
    }
    for(i = 0; i < C2_LENGTH; i++){
        for(j = 0; j < C2_DIMENSIONS; j++){
            for(k = 0; k < C2_DIMENSIONS; k++){
                network->C2_Images->image[i][j][k] = 0;
            }
        }
    }
    for(i = 0; i < S2_LENGTH; i++){
        for(j = 0; j < S2_DIMENSIONS; j++){
            for(k = 0; k < S2_DIMENSIONS; k++){
                network->S2_Images->image[i][j][k] = 0;
            }
        }
    }
    for(i = 0; i < S2_LENGTH * S2_DIMENSIONS * S2_DIMENSIONS; i++){
        network->S2_Vector[i] = 0;
    }
    for(i = 0; i < OUTPUT_LENGTH; i++){
        network->calculated_output[i] = 0;
    }
    for(i = 0; i < OUTPUT_LENGTH; i++){
        network->desired_output[i] = 0;
    }
}

//Sets all parameters within a network to 0
void zero_parameters(cnn* network){
    int i, n, j, k;
    
    for(i = 0; i < C1_LENGTH; i++){
        for(j = 0; j < KERNEL_SIZE; j++){
            for(k = 0; k < KERNEL_SIZE; k++){
                network->C1_Kernels[0][i][j][k] = 0;
            }
        }
    }
    for(i = 0; i < S1_LENGTH; i++){
        for(n = 0; n < C2_LENGTH; n++){
            for(j = 0; j < KERNEL_SIZE; j++){
                for(k = 0; k < KERNEL_SIZE; k++){
                    network->C2_Kernels[i][n][j][k] = 0;
                }
            }
        }
    }
    for(j = 0; j < OUTPUT_LENGTH; j++){
        for(k = 0; k < S2_LENGTH * S2_DIMENSIONS * S2_DIMENSIONS; k++){
            network->output_weights[j][k] = 0;
        }
    }
    for(i = 0; i < C1_LENGTH; i++){
        network->C1_Biases[i] = 0;
    }
    for(i = 0; i < C2_LENGTH; i++){
        network->C2_Biases[i] = 0;
    }
    for(i = 0; i < OUTPUT_LENGTH; i++){
        network->output_biases[i] = 0;
    }
    
}

//Generates a uniform random digit within a specified range
double uniform_rand(){
    double value = (double)rand()/RAND_MAX;
    value *= (UPPER - LOWER);
    value += LOWER;
    return value;
}

//Reads header items from input files
void read_headers(FILE *image_ptr, FILE *label_ptr){
    uint32_t label_magic_number = 0u;
    label_magic_number |= getc(label_ptr) << 24;
    label_magic_number |= getc(label_ptr) << 16;
    label_magic_number |= getc(label_ptr) << 8;
    label_magic_number |= getc(label_ptr);

    //Item count from label file
    uint32_t label_item_count = 0u;
    label_item_count |= getc(label_ptr) << 24;
    label_item_count |= getc(label_ptr) << 16;
    label_item_count |= getc(label_ptr) << 8;
    label_item_count |= getc(label_ptr);

    //Magic Number from images file
    uint32_t images_magic_number = 0u;
    images_magic_number |= getc(image_ptr) << 24;
    images_magic_number |= getc(image_ptr) << 16;
    images_magic_number |= getc(image_ptr) << 8;
    images_magic_number |= getc(image_ptr);

    //Item count from images file
    uint32_t images_item_count = 0u;
    images_item_count |= getc(image_ptr) << 24;
    images_item_count |= getc(image_ptr) << 16;
    images_item_count |= getc(image_ptr) << 8;
    images_item_count |= getc(image_ptr);

    //Rows per image
    uint32_t rows = 0u;
    rows |= getc(image_ptr) << 24;
    rows |= getc(image_ptr) << 16;
    rows |= getc(image_ptr) << 8;
    rows |= getc(image_ptr);

    //Columns per image
    uint32_t columns = 0u;
    columns |= getc(image_ptr) << 24;
    columns |= getc(image_ptr) << 16;
    columns |= getc(image_ptr) << 8;
    columns |= getc(image_ptr);
    // printf("Label magic: %u\n",label_magic_number);
    // printf("Label count: %u\n",label_item_count);
    // printf("Image magic: %u\n",images_magic_number);
    // printf("Image count: %u\n",images_item_count);
}

//Reads in an image and it's label from the the input files
void load_image(cnn* network, FILE *image_ptr, FILE *label_ptr){
    int j, k;
    double value = 0;
    for(j = 0; j < INPUT_DIMENSIONS; j++){
        for(k = 0; k < INPUT_DIMENSIONS; k++){
#ifdef USE_RANDOM
            value = fRand();
#else
            value = (double)getc(image_ptr);
            value /= 255;
#endif
            network->input_image[j][k] = value;
        }
    }
#ifdef USE_RANDOM
    double t = (double)(random()%10);
#else
    double t = (double)getc(label_ptr);
#endif
    for(j = 0; j < 10; j++){
        if((int)t == j)
            network->desired_output[j] = 1;
        else
            network->desired_output[j] = 0;
        //printf("%lf\n",network->desired_output[j]);
    }
    //print_image(network->input_image, 28);
}


void load_C1_gpu(cnn* network){
	int i,j,k, errCount;
	hipEvent_t start, stop;
	hipEvent_t start1, stop1;
	srand(time(NULL));
    	struct timespec time_start, time_stop;
	float elapsed_gpu, elapsed_gpu1;
	double elapsed_cpu, error, max_error;
	double* d_output_images;
	double* d_input_image;
	double* d_kernels;
	double* d_biases;
	image_vector* h_img;
        dim3 my_block(C1_DIMENSIONS, C1_DIMENSIONS); // 24x24 computation can be done in parallel - each element will be computed using one thread
        dim3 my_grid(1, C1_LENGTH); // 6 blocks for 6 channels
        errCount = 0;
        max_error = 0.0;

	CUDA_SAFE_CALL(hipSetDevice(0));

#ifdef PRINT_TIME1
        // Create the cuda events
        hipEventCreate(&start1);
        hipEventCreate(&stop1);
        // Record event on the default stream
        hipEventRecord(start1, 0);
#endif

	d_input_image = cu_memcpy_array(network->input_image, INPUT_DIMENSIONS, INPUT_DIMENSIONS);
	d_output_images = cu_malloc_images(C1_DIMENSIONS, C1_LENGTH);
	d_kernels = cu_memcpy_kernel(network->C1_Kernels, 1, C1_LENGTH, KERNEL_SIZE); 
	d_biases = cu_memcpy_vector(network->C1_Biases, C1_LENGTH); 
	//dim3 my_block(C1_DIMENSIONS, C1_DIMENSIONS); // 24x24 computation can be done in parallel - each element will be computed using one thread
  	//dim3 my_grid(1, C1_LENGTH); // 6 blocks for 6 channels
	//errCount = 0;
	//max_error = 0.0;

#ifdef PRINT_TIME
  	// Create the cuda events
  	hipEventCreate(&start);
  	hipEventCreate(&stop);
  	// Record event on the default stream
  	hipEventRecord(start, 0);
#endif

	c1_kernel<<<my_grid, my_block>>>(d_input_image, d_kernels, d_biases, d_output_images);


#ifdef PRINT_TIME
  	// Stop and destroy the timer
  	hipEventRecord(stop,0);
  	hipEventSynchronize(stop);
  	hipEventElapsedTime(&elapsed_gpu, start, stop);
  	//printf("\nGPU time: %f (msec)\n", elapsed_gpu);
  	hipEventDestroy(start);
  	hipEventDestroy(stop);
#endif

  	// Check for errors during launch
  	CUDA_SAFE_CALL(hipPeekAtLastError());

	//functionality check
	h_img = h_memcpy_image(d_output_images, C1_DIMENSIONS, C1_LENGTH);
	
#ifdef PRINT_TIME1
        // Stop and destroy the timer
        hipEventRecord(stop1,0);
        hipEventSynchronize(stop1);
        hipEventElapsedTime(&elapsed_gpu1, start1, stop1);
        //printf("\nGPU time with mem: %f (msec)\n", elapsed_gpu1);
        hipEventDestroy(start1);
        hipEventDestroy(stop1);
#endif

	clock_gettime(CLOCK_REALTIME, &time_start);
	load_C1(network);
	clock_gettime(CLOCK_REALTIME, &time_stop);
	elapsed_cpu = interval(time_start, time_stop);

    	for(i = 0; i < C1_LENGTH; i++){
    		for(j = 0; j < C1_DIMENSIONS; j++){
        		for(k = 0; k < C1_DIMENSIONS; k++){
                		error = fabsf(network->C1_Images->image[i][j][k] - h_img->image[i][j][k]) ;
				//printf("ERROR: %lf\n",h_img->image[i][j][k]);
				//printf("GOLDE: %lf\n",network->C1_Images->image[i][j][k]);
    				if ( error > fabsf(network->C1_Images->image[i][j][k]*0.05) ) { // Checking errors with 5% tolerence
      					errCount++;
    				}
    				if (error>max_error) {
      					max_error = error;
    				}
			}
		}
	}
	printf("CPU: %lfms\t, GPU: %fms\tGPU with mem: %fms\t\n",(elapsed_cpu*1000),elapsed_gpu,elapsed_gpu1);
	//printf("CPU: %lfms\t, GPU: %fms\t, ErrorCount: %d\t, MaxError: %lf\n",(elapsed_cpu*1000),elapsed_gpu,errCount,max_error);
	
}



//Propagate from input to C1
void load_C1(cnn* network){
    //print_image(network->input_image, 28);
    //Create a list of the KERNEL_SIZExKERNEL_SIZE sections of the input image
    int length = C1_DIMENSIONS * C1_DIMENSIONS;
    image_vector* input_sections = new_image_vector(KERNEL_SIZE, length);
    int i, j, k;
    for(i = 0; i < length; i++){
        for(j = 0; j < KERNEL_SIZE; j++){
            for(k = 0; k < KERNEL_SIZE; k++){
                input_sections->image[i][j][k] = network->input_image[(int)floor((double)i/C1_DIMENSIONS) + j][i%C1_DIMENSIONS + k];
            }
        }
        //print_image(input_sections->image[i], 5);
    }
    for(i = 0; i < C1_LENGTH; i++){
        for(j = 0; j < C1_DIMENSIONS; j++){
            for(k = 0; k < C1_DIMENSIONS; k++){
                network->C1_Images->image[i][j][k] = activation(
                        dot_product(input_sections->image[(j*C1_DIMENSIONS)+k], network->C1_Kernels[0][i], KERNEL_SIZE)
                        + network->C1_Biases[i]);
            }
        }
        //print_image(network->C1_Images->image[i], 24);
    }
    free_image_vector(input_sections, KERNEL_SIZE, length);
    free(input_sections);
}

void load_S1(cnn* network){
    int i, n, j, k;
    image_vector* C1_sections = new_image_vector(2, S1_DIMENSIONS*S1_DIMENSIONS);
    //print_image(network->C1_Images->image[0], 24);
    for(i = 0; i < S1_LENGTH; i++){
        for(n = 0; n < 2 * S1_DIMENSIONS*S1_DIMENSIONS; n++){
            if(n%2 == 0){
                for(j = 0; j < 2; j++){
                    for(k = 0; k < 2; k++){
                        C1_sections->image[n/2][j][k] = 
                                network->C1_Images->image[i][(int)floor((double)n/C1_DIMENSIONS)*2 + j][(n%C1_DIMENSIONS) + k];
                    }
                }
                //if(i == 0)
                    //print_image(C1_sections->image[n/2], 2);
            }
        }
        
        for(j = 0; j < S1_DIMENSIONS; j++){
            for(k = 0; k < S1_DIMENSIONS; k++){
                network->S1_Images->image[i][j][k] = average_matrix(C1_sections->image[(j*12) + k], 2);
            }
        }
    }
    //print_image(network->S1_Images->image[0], 12);
    free_image_vector(C1_sections, 2, S1_DIMENSIONS*S1_DIMENSIONS);
    free(C1_sections);
}

void load_C2(cnn* network){
    image_vector* S1_sections[S1_LENGTH];
    int l, m, n, i, j, k;
    int length = C2_DIMENSIONS * C2_DIMENSIONS;
    for(i = 0; i < S1_LENGTH; i++){
        S1_sections[i] = new_image_vector(KERNEL_SIZE, length);
    }
    //print_image(network->S1_Images->image[0], 12);
    for(n = 0; n < S1_LENGTH; n++){
        for(i = 0; i < length; i++){
            for(j = 0; j < KERNEL_SIZE; j++){
                for(k = 0; k < KERNEL_SIZE; k++){
                    S1_sections[n]->image[i][j][k] =
                            network->S1_Images->image[n][(int)floor((double)i/C2_DIMENSIONS) + j][i%C2_DIMENSIONS + k];
                }
            }
            //if(n < 2)
                //print_image(S1_sections[n]->image[i], 5);
        }
    }
    
    for(m = 0; m < C2_LENGTH; m++){
        for(n = 0; n < C2_DIMENSIONS; n++){
            for(i = 0; i < C2_DIMENSIONS; i++){
                for(l = 0; l < S1_LENGTH; l++){
                    network->C2_Images->image[m][n][i] +=
                            dot_product(S1_sections[l]->image[(n*C2_DIMENSIONS) + i], network->C2_Kernels[l][m], KERNEL_SIZE);
                }
            }
        }
    }
    
    for(i = 0; i < S1_LENGTH; i++){
        free_image_vector(S1_sections[i], KERNEL_SIZE, length);
    }
    free(*S1_sections);
}

void load_S2(cnn* network){
    int i, n, j, k;
    image_vector* C2_sections = new_image_vector(2, S2_DIMENSIONS * S2_DIMENSIONS);
    //print_image(network->C2_Images->image[0],8);
    for(i = 0; i < C2_LENGTH; i++){
        for(n = 0; n < 2 * S2_DIMENSIONS * S2_DIMENSIONS; n++){
            if(n%2 == 0){
                for(j = 0; j < 2; j++){
                    for(k = 0; k < 2; k++){
                        C2_sections->image[n/2][j][k] = 
                                network->C2_Images->image[i][(int)floor((double)n/C2_DIMENSIONS)*2 + j][n%C2_DIMENSIONS + k];
                    }
                }
                //if(i == 0)
                    //print_image(C2_sections->image[n/2], 2);
            }
        }
        
        for(j = 0; j < S2_DIMENSIONS; j++){
            for(k = 0; k < S2_DIMENSIONS; k++){
                network->S2_Images->image[i][j][k] = average_matrix(C2_sections->image[(j*S2_DIMENSIONS)+k],2);
            }
        }
        //print_image(network->S2_Images->image[i], 4);
    }
    free_image_vector(C2_sections, 2, S2_DIMENSIONS * S2_DIMENSIONS);
}

void concatenate_S2(cnn* network){
    //print_image(network->S2_Images->image[0],4);
    int n, j, k;
    for(n = 0; n < S2_LENGTH; n++){
        for(j = 0; j < S2_DIMENSIONS; j ++){
            for(k = 0; k < S2_DIMENSIONS; k++){
                network->S2_Vector[(n*(S2_DIMENSIONS*S2_DIMENSIONS)) + (j*S2_DIMENSIONS) + k] = network->S2_Images->image[n][j][k];
                //printf("%lf\n",network->S2_Vector[(n*(S2_DIMENSIONS*S2_DIMENSIONS)) + (j*S2_DIMENSIONS) + k]);
            }
        }
    }
}

void load_output(cnn* network){
    int i, n, j, k;
    double value = 0;
    for(i = 0; i < OUTPUT_LENGTH; i++){
        for(n = 0; n < S2_LENGTH * S2_DIMENSIONS * S2_DIMENSIONS; n++){
            value += (network->S2_Vector[n] * network->output_weights[i][n]);
        }
        value += network->output_biases[i];
        network->calculated_output[i] = output_activation(value);
        value = 0;
    }
}

void backpropagation(cnn* network, cnn* gradient){
    int i, n, u, v, j, k, l, q;
    double sum = 0;
    //Delta W
    #pragma omp parallel for private(i,j,sum) 
    for(i = 0; i < OUTPUT_LENGTH; i++){
        for(j = 0; j < S2_LENGTH * S2_DIMENSIONS * S2_DIMENSIONS; j++){
             sum += (network->calculated_output[i] - network->desired_output[i]) * output_activation_derivative(network->calculated_output[i]) * network->S2_Vector[j];
            gradient->output_weights[i][j] += sum;
        }
    }
    sum = 0;
    //Delta B
    #pragma omp parallel for private(i,sum) 
    for(i = 0; i < OUTPUT_LENGTH; i++){
        sum += (network->calculated_output[i] - network->desired_output[i])
                * output_activation_derivative(network->calculated_output[i]);
        gradient->output_biases[i] += sum;
    }
    sum = 0;
    //Delta f(j)
    #pragma omp parallel for private(i,j,sum) 
    for(j = 0; j < S2_LENGTH * S2_DIMENSIONS * S2_DIMENSIONS; j++){
        for(i = 0; i < OUTPUT_LENGTH; i++){
            sum += (gradient->output_biases[i] * network->output_weights[i][j]);
        }
        gradient->S2_Vector[j] += sum;
    }
    //un-concatenate
    #pragma omp parallel for private(n,k,j) 
    for(n = 0; n < S2_LENGTH; n++){
        for(j = 0; j < S2_DIMENSIONS; j++){
            for(k = 0; k < S2_DIMENSIONS; k++){
                gradient->S2_Images->image[n][j][k] = gradient->S2_Vector[(n*S2_DIMENSIONS*S2_DIMENSIONS) + (j*S2_DIMENSIONS) + k];
            }
        }
    }
    //super sample
    #pragma omp parallel for private(n,k,j) 
    for(n = 0; n < C2_LENGTH; n++){
        for(j = 0; j < C2_DIMENSIONS; j++){
            for(k = 0; k < C2_DIMENSIONS; k++){
                gradient->C2_Images->image[n][j][k] = (0.25) * gradient->S2_Images->image[n][(int)floor((double)j/2)][(int)floor((double)k/2)];
            }
        }
    }
    sum = 0;
    //Delta K2
    #pragma omp parallel for private(n,i,u,v,j,k,sum) 
    for(n = 0; n < S1_LENGTH; n++){
        for(i = 0; i < C2_LENGTH; i++){
            for(u = 0; u < KERNEL_SIZE; u++){
                for(v = 0; v < KERNEL_SIZE; v++){
                    for(j = 0; j < C2_DIMENSIONS; j++){
                        sum = 0;
                        for(k = 0; k < C2_DIMENSIONS; k++){
                            sum += gradient->C2_Images->image[i][j][k] *
                                    activation_derivative(network->C2_Images->image[i][j][k]) *
                                    network->S1_Images->image[n][j+u][k+v];
                            
                        }
                        gradient->C2_Kernels[n][i][u][v] += sum;
                    }
                }
            }
        }
    }
    //Delta B2
    #pragma omp parallel for private(n,j,k,sum)
    for(n = 0; n < C2_LENGTH; n++){
        sum = 0;
        for(j = 0; j < C2_DIMENSIONS; j++){
            for(k = 0; k < C2_DIMENSIONS; k++){
                sum += gradient->C2_Images->image[n][j][k] *
                        activation_derivative(network->C2_Images->image[n][j][k]);
            }
        }
        gradient->C2_Biases[n] += sum;
    }
    //Funniest shit i've pulled so consider in debugging
    //Prep for backprop convolution
    image_vector* C2_gradient_temp = new_image_vector(S1_DIMENSIONS + KERNEL_SIZE - 1, C2_LENGTH);
    image_vector* C2_network_temp = new_image_vector(S1_DIMENSIONS + KERNEL_SIZE - 1, C2_LENGTH);
    #pragma omp parallel for private(n,j,k)
    for(i = 0; i < C2_LENGTH; i++){
        for(j = 0; j < S1_DIMENSIONS + KERNEL_SIZE - 1; j++){
            for(k = 0; k < S1_DIMENSIONS + KERNEL_SIZE - 1; k++){
                C2_gradient_temp->image[i][j][k] = 0;
                C2_network_temp->image[i][j][k] = 0;
            }
        }
    }
    #pragma omp parallel for private(i,j,k)
    for(i = 0; i < C2_LENGTH; i++){
        for(j = 4; j < C2_DIMENSIONS + 4; j++){
            for(k = 4; k < C2_DIMENSIONS + 4; k++){
                C2_gradient_temp->image[i][j][k] = gradient->C2_Images->image[i][j-4][k-4];
                C2_network_temp->image[i][j][k] = network->C2_Images->image[i][j-4][k-4];
            }
        }
    }
    //Delta Sp1
    #pragma omp parallel for private(n,i,l,q,u,v,sum)
    for(n = 0; n < S1_LENGTH; n++){
        for(i = 0; i < S1_DIMENSIONS; i++){
            for(l = 0; l < S1_DIMENSIONS; l++){
                for(q = 0; q < C2_LENGTH; q++){
                    sum = 0;
                    for(u = 0; u < KERNEL_SIZE; u++){
                        for(v = 0; v < KERNEL_SIZE; v++){
                            sum +=  C2_gradient_temp->image[q][i+u][l+v] *
                                    activation_derivative(C2_network_temp->image[q][i+u][l+v]) *
                                    network->C2_Kernels[n][q][u][v];
                        }
                    }  
                    gradient->S1_Images->image[n][i][l] += sum;      
                }
            }
        }   
    }
    free_image_vector(C2_gradient_temp, S1_DIMENSIONS + KERNEL_SIZE - 1, C2_LENGTH);
    free_image_vector(C2_network_temp, S1_DIMENSIONS + KERNEL_SIZE - 1, C2_LENGTH);
    free(C2_gradient_temp);
    free(C2_network_temp);
    
    //super sampling
    #pragma omp parallel for private(n,j,k)
    for(n = 0; n < C1_LENGTH; n++){
        for(j = 0; j < C1_DIMENSIONS; j++){
            for(k = 0; k < C1_DIMENSIONS; k++){
                gradient->C1_Images->image[n][j][k] = 
                        (0.25) * gradient->S1_Images->image[n][(int)floor((double)j/2)][(int)floor((double)k/2)];
            }
        }
    }
    //Delta K1
    #pragma omp parallel for private(n,u,v,j,k)
    for(n = 0; n < C1_LENGTH; n++){
        for(u = 0; u < KERNEL_SIZE; u++){
            for(v = 0; v < KERNEL_SIZE; v++){
                sum = 0;
                for(j = 0; j < C1_DIMENSIONS; j++){
                    for(k = 0; k < C1_DIMENSIONS; k++){
                        sum +=  gradient->C1_Images->image[n][j][k] *
                                activation_derivative(network->C1_Images->image[n][j][k]) *
                                network->input_image[j+u][k+v];
                    }
                }
                gradient->C1_Kernels[0][n][u][v] += sum;
            }
        }
    }
    #pragma omp parallel for private(n,j,k)
    for(n = 0; n < C1_LENGTH; n++){
        sum = 0;
        for(j = 0; j < C1_DIMENSIONS; j++){
            for(k = 0; k < C1_DIMENSIONS; k++){
                sum += gradient->C1_Images->image[n][j][k];
            }
        }
        gradient->C1_Biases[n] += sum;
    } 
}

void update_batch_gradient(cnn* image_gradient, cnn* batch_gradient){
    int n, m, j, k;
    for(n = 0; n < C1_LENGTH; n++){
        for(j = 0; j < KERNEL_SIZE; j++){
            for(k = 0; k < KERNEL_SIZE; k++){
                batch_gradient->C1_Kernels[0][n][j][k] +=
                        image_gradient->C1_Kernels[0][n][j][k];
            }
        }
        batch_gradient->C1_Biases[n] += image_gradient->C1_Biases[n];
    }
    for(n = 0; n < S1_LENGTH; n++){
        for(m = 0; m < C2_LENGTH; m++){
            for(j = 0; j < KERNEL_SIZE; j++){
                for(k = 0; k < KERNEL_SIZE; k++){
                    batch_gradient->C2_Kernels[n][m][j][k] +=
                            image_gradient->C2_Kernels[n][m][j][k];
                }
            }
            batch_gradient->C2_Biases[m] +=
                    image_gradient->C2_Biases[m];
        }
    }
    for(n = 0; n < OUTPUT_LENGTH; n++){
        for(m = 0; m < S2_LENGTH * S2_DIMENSIONS * S2_DIMENSIONS; m++){
            batch_gradient->output_weights[n][m] +=
                    image_gradient->output_weights[n][m];
            
        }
        batch_gradient->output_biases[n] +=
                image_gradient->output_biases[n];
    }
}

void gradient_descent(cnn* network, cnn* gradient){
    int n, m, j, k;
    for(n = 0; n < C1_LENGTH; n++){
        for(j = 0; j < KERNEL_SIZE; j++){
            for(k = 0; k < KERNEL_SIZE; k++){
                network->C1_Kernels[0][n][j][k] -=
                        LEARNING_RATE*(gradient->C1_Kernels[0][n][j][k]/BATCH_SIZE);
            }
        }
        network->C1_Biases[n] -= LEARNING_RATE*(gradient->C1_Biases[n]/BATCH_SIZE);
    }
    for(n = 0; n < S1_LENGTH; n++){
        for(m = 0; m < C2_LENGTH; m++){
            for(j = 0; j < KERNEL_SIZE; j++){
                for(k = 0; k < KERNEL_SIZE; k++){
                    network->C2_Kernels[n][m][j][k] -=
                            LEARNING_RATE*(gradient->C2_Kernels[n][m][j][k]/BATCH_SIZE);
                }
            }
            network->C2_Biases[m] -=
                    LEARNING_RATE*(gradient->C2_Biases[m]/BATCH_SIZE);
        }
    }
    for(n = 0; n < OUTPUT_LENGTH; n++){
        for(m = 0; m < S2_LENGTH * S2_DIMENSIONS * S2_DIMENSIONS; m++){
            network->output_weights[n][m] -=
                    LEARNING_RATE*(gradient->output_weights[n][m]/BATCH_SIZE);
            
        }
        network->output_biases[n] -=
                LEARNING_RATE*(gradient->output_biases[n]/BATCH_SIZE);
    }
}

void free_image_vector(image_vector* images, int dimensions, int length){
    int i;
    for(i = 0; i < length; i++){
        free_array(images->image[i], dimensions);
        //free(images->image[i]); //chath removed double free
    }
    free(images->image);
}

void free_array(array array, int rows){
    int j, k;
    for(j = 0; j < rows; j++){
        free(array[j]);
    }
    free(array);
}

//Computes the average value of a matrix
double average_matrix(array image, int dimensions){
    int j, k;
    double value = 0;
    for(j = 0; j < dimensions; j++){
        for(k = 0; k < dimensions; k++){
            value += image[j][k];
        }
    }
    value /= (dimensions * dimensions);
    return value;
}

void print_image(array image, int dimensions){
    int j, k;
    for(j = 0; j < dimensions; j++){
        for(k = 0; k < dimensions; k++){
            if(image[j][k] > 0)
                printf(RED"[%.2lf]",image[j][k]);
            else
                printf(RESET"[%.2lf]",image[j][k]);
        }
        printf(RESET"\n");
    }
    printf(RESET"\n");
}

//Computes the dot product of two identically sized matrices
double dot_product(array array1, array array2, int dimensions){
    int j, k;
    double value = 0;
    for(j = 0; j < dimensions; j++){
        for(k = 0; k < dimensions; k++){
            value += array1[j][k] * array2[j][k];
        }
    }
    return value;
}

double activation(double x){
    if(x < 0)
        return 0.01*x;
    return x;
}

double activation_derivative(double x){
    if(x < 0)
        return 0.01;
    return 1;
}

double output_activation(double x){
    return (double)1/(1+exp(-x));
}

double output_activation_derivative(double x){
    return x * (1 - x);
}

double loss_function(cnn* network){
    int i;
    double value = 0;
    for(i = 0; i < 10; i++){
        value += (network->calculated_output[i] - network->desired_output[i])
                * (network->calculated_output[i] - network->desired_output[i]);
        //printf("%lf\n",network->calculated_output[i]);
                
    }
    return value/2;
}
